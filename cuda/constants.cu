#include "cuda_utils.h"

__device__ __constant__ int elements;

int main() {
  int elems = 1;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("elements"), &elems, sizeof(int)));
}
