
#include <hip/hip_runtime.h>
__global__ void vec_add(float *a, float *b, float *c, int N) {
  int idx = threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx]+b[idx];
  }
}

