#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define NUM_ELEMS 10

/**
  This function will add the vectors a and b together and save it into N
  */
extern __global__ void vec_add(float *a, float *b, float *c, int n);

int main() {
  // cuda specific configs
  int blocksize = NUM_ELEMS;
  int numblocks = 1;

  // declaring variables
  float *a,*b,*c;
  a = (float*)malloc(sizeof(float)*NUM_ELEMS);
  b = (float*)malloc(sizeof(float)*NUM_ELEMS);
  c = (float*)calloc(sizeof(float),NUM_ELEMS);


  // fill variables with data
  for (int i=0;i<NUM_ELEMS;i++) {
    a[i] = (float) i;
    b[i] = ((float) NUM_ELEMS-i);
  }

  // allocate device memory
  float *d_a, *d_b, *d_c;  
  hipMalloc((void**)&d_a,sizeof(float)*NUM_ELEMS);
  hipMalloc((void**)&d_b,sizeof(float)*NUM_ELEMS);
  hipMalloc((void**)&d_c,sizeof(float)*NUM_ELEMS);

  // copy data from host to device
  hipMemcpy(d_a,a,NUM_ELEMS*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,NUM_ELEMS*sizeof(float), hipMemcpyHostToDevice);
  
  // launch kernel
  dim3 dimBlock(blocksize);
  dim3 dimGrid(numblocks);
  vec_add<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,NUM_ELEMS);
  
  hipDeviceSynchronize();
  // copy data from device back to host
  hipMemcpy(c,d_c,NUM_ELEMS*sizeof(float),hipMemcpyDeviceToHost);

  // print everything
  printf("Finished, printing data\n");
  for (int i=0;i<NUM_ELEMS;i++) {
    printf("%f = %f + %f\n",c[i],a[i],b[i]);
  }

  return 0;
}
